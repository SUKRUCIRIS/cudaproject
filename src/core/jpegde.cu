#include "jpegde.cuh"
#include "utility.cuh"
// ŞÜKRÜ ÇİRİŞ 2024

SKR::jpegde::jpegde(void)
{
    CHECK_NVJPEG(nvjpegCreateSimple(&handle))
    CHECK_NVJPEG(nvjpegJpegStateCreate(handle, &state))
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_NVJPEG(nvjpegEncoderStateCreate(handle, &enc_state, stream));
    CHECK_NVJPEG(nvjpegEncoderParamsCreate(handle, &enc_params, stream));
}

SKR::jpegde &SKR::jpegde::getInstance()
{
    static jpegde ins;
    return ins;
}

std::vector<unsigned char> *SKR::jpegde::readJPEG(const std::string &filename)
{
    MEASURE_TIME1;
    std::vector<unsigned char> *buffer = new std::vector<unsigned char>;
    std::ifstream file(filename, std::ios::in | std::ios::binary);
    if (!file)
    {
        std::cout << "Failed to open file for reading: " << filename << std::endl;
        exit(-1);
    }

    file.seekg(0, std::ios::end);
    std::streamsize size = file.tellg();
    file.seekg(0, std::ios::beg);

    buffer->resize(size);
    if (!file.read(reinterpret_cast<char *>(buffer->data()), size))
    {
        std::cout << "Failed to read the file: " << filename << std::endl;
        exit(-1);
    }
    MEASURE_TIME2("readJPEG");
    return buffer;
}

SKR::Image *SKR::jpegde::decodeJPEG(const std::vector<unsigned char> &jpeg_buffer)
{
    MEASURE_TIME1;
    int nComponents;
    int widths[NVJPEG_MAX_COMPONENT];
    int heights[NVJPEG_MAX_COMPONENT];

    Image *output_image = new Image;

    nvjpegChromaSubsampling_t subsampling;

    CHECK_NVJPEG(nvjpegGetImageInfo(handle, jpeg_buffer.data(), jpeg_buffer.size(), &nComponents, &subsampling, widths, heights));

    output_image->width = widths[0];
    output_image->height = heights[0];
    int size = heights[0] * widths[0];
    for (int i = 0; i < nComponents; i++)
    {
        CHECK_CUDA(hipMalloc(&output_image->image.channel[i], size));
        output_image->image.pitch[i] = widths[0];
    }
    for (int i = nComponents; i < NVJPEG_MAX_COMPONENT; i++)
    {
        output_image->image.channel[i] = 0;
        output_image->image.pitch[i] = 0;
    }

    CHECK_NVJPEG(nvjpegDecode(handle, state, jpeg_buffer.data(), jpeg_buffer.size(), NVJPEG_OUTPUT_RGB, &output_image->image, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));
    MEASURE_TIME2("decodeJPEG");
    return output_image;
}

SKR::jpegde::~jpegde()
{
    CHECK_NVJPEG(nvjpegEncoderParamsDestroy(enc_params));
    CHECK_NVJPEG(nvjpegEncoderStateDestroy(enc_state));
    CHECK_NVJPEG(nvjpegJpegStateDestroy(state));
    CHECK_NVJPEG(nvjpegDestroy(handle));
    CHECK_CUDA(hipStreamDestroy(stream));
}

void SKR::jpegde::freeJPEG(Image *image)
{
    MEASURE_TIME1;
    for (int i = 0; i < NVJPEG_MAX_COMPONENT; i++)
    {
        CHECK_CUDA(hipFree(image->image.channel[i]));
    }

    delete image;
    MEASURE_TIME2("freeJPEG");
}

std::vector<unsigned char> *SKR::jpegde::encodeJPEG(const Image *image, const int quality, const bool isgray)
{
    MEASURE_TIME1;
    if (isgray)
    {
        CHECK_NVJPEG(nvjpegEncoderParamsSetSamplingFactors(enc_params, NVJPEG_CSS_GRAY, stream));
    }
    else
    {
        CHECK_NVJPEG(nvjpegEncoderParamsSetSamplingFactors(enc_params, NVJPEG_CSS_420, stream));
    }
    CHECK_NVJPEG(nvjpegEncoderParamsSetQuality(enc_params, quality, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    CHECK_NVJPEG(nvjpegEncodeImage(handle, enc_state, enc_params, &(image->image),
                                   NVJPEG_INPUT_RGB, image->width, image->height, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    size_t length = 0;
    CHECK_NVJPEG(nvjpegEncodeRetrieveBitstream(handle, enc_state, 0, &length, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    std::vector<unsigned char> *encoded = new std::vector<unsigned char>(length);
    CHECK_NVJPEG(nvjpegEncodeRetrieveBitstream(handle, enc_state, encoded->data(), &length, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));
    MEASURE_TIME2("encodeJPEG");
    return encoded;
}

void SKR::jpegde::writeJPEG(const std::string &filename, const std::vector<unsigned char> &jpeg_buffer)
{
    MEASURE_TIME1;
    std::ofstream file(filename, std::ios::out | std::ios::binary);
    if (!file)
    {
        std::cout << "Failed to open file for writing: " << filename << std::endl;
        exit(-1);
    }
    file.write(reinterpret_cast<const char *>(jpeg_buffer.data()), jpeg_buffer.size());
    if (!file)
    {
        std::cout << "Failed to write data to file: " << filename << std::endl;
        exit(-1);
    }
    file.close();
    MEASURE_TIME2("writeJPEG");
}