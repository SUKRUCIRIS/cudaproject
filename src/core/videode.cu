#include "videode.cuh"
#include "utility.cuh"

SKR::videode::videode()
{
    CHECK_NVDEC(hipCtxCreate(&cuContext, 0, 0));
}

SKR::videode::~videode()
{
    CHECK_NVDEC(hipCtxDestroy(cuContext));
}

SKR::videode &SKR::videode::getInstance()
{
    static videode ins;
    return ins;
}